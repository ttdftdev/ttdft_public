#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2020-2021.                                                   *
 * The Regents of the University of Michigan and TTDFT authors.               *
 *                                                                            *
 * This file is part of the TTDFT code.                                       *
 *                                                                            *
 * TTDFT is free software: you can redistribute it and/or modify              *
 *  it under the terms of the Lesser GNU General Public License as            *
 *  published by the Free Software Foundation, either version 3 of            *
 *  the License, or (at your option) any later version.                       *
 *                                                                            *
 *  TTDFT is distributed in the hope that it will be useful, but              *
 *  WITHOUT ANY WARRANTY; without even the implied warranty                   *
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.                      *
 *  See the Lesser GNU General Public License for more details.               *
 *                                                                            *
 * You should have received a copy of the GNU Lesser General Public           *
 * License at the top level of TTDFT distribution.  If not, see               *
 * <https://www.gnu.org/licenses/>.                                           *
 ******************************************************************************/

#include <iostream>
#include <vector>
#include <hipsparse.h>
#include <hipblas.h>
#include "Mult.h"

namespace {
    __global__ void adjust_column(int owned_row_start,
                                  int owned_row_size,
                                  int n,
                                  int *col_idx);
    extern "C" {

    }
}

AX::AX(int num_bg,
       int num_wfns) : double_one(1.0),
                       double_zero(0.0),
                       num_wfns(num_wfns),
                       num_band_groups(num_bg),
                       device_comm(num_bg),
                       owned_row_start(0),
                       owned_row_end(0),
                       owned_cpu_start(0),
                       owned_cpu_end(0),
                       owned_wfn_idx(num_bg + 1,
                                     0),
                       owned_wfn_start(0),
                       owned_wfn_end(0) {

    hipsparseStatus_t sparse_status;
    if (device_comm.owned_device) {
        hipsparseCreate(&sparse_handle);
        sparse_status = hipsparseCreateMatDescr(&sparse_descr);
        if (sparse_status != HIPSPARSE_STATUS_SUCCESS) {
            std::cout << "Matrix descriptor initialization failed" << std::endl;
            std::terminate();
        }
        hipsparseSetMatType(sparse_descr,
                           HIPSPARSE_MATRIX_TYPE_GENERAL);
        hipsparseSetMatIndexBase(sparse_descr,
                                HIPSPARSE_INDEX_BASE_ZERO);

        hipblasStatus_t blas_stat;
        blas_stat = hipblasCreate(&blas_handle);
        if (blas_stat != HIPBLAS_STATUS_SUCCESS) {
            std::cout << "CUBLAS Library create handle failed" << std::endl;
            std::terminate();
        }
    }

    std::vector<int> owned_wfns(num_band_groups,
                                num_wfns / num_band_groups);
    for (int i = 0; i < num_wfns % num_band_groups; ++i) {
        owned_wfns[i] += 1;
    }
    for (int i = 0; i < num_band_groups; ++i) {
        owned_wfn_idx[i + 1] = owned_wfn_idx[i] + owned_wfns[i];
    }
    if (device_comm.owned_band_block != MPI_UNDEFINED) {
        owned_wfn_start = owned_wfn_idx[device_comm.owned_band_block];
        owned_wfn_end = owned_wfn_idx[device_comm.owned_band_block + 1];
    }

    int owned_col_size = 0;
    if (device_comm.owned_device) {
        owned_col_size = owned_wfns[device_comm.owned_band_block];
    }
    std::vector<PetscInt> wfn_col_idx_temp(owned_col_size,
                                           owned_wfn_start);
    for (int i = 0; i < wfn_col_idx_temp.size(); ++i) {
        wfn_col_idx_temp[i] += i;
    }
    PetscInt wfn_col_idx_size = owned_wfn_end - owned_wfn_start;
    ISCreateGeneral(MPI_COMM_WORLD,
                    wfn_col_idx_size,
                    wfn_col_idx_temp.data(),
                    PETSC_COPY_VALUES,
                    &wfn_col_is);

}

void AX::setup_system(Mat A) {
    PetscInt global_am, global_an;
    MatGetSize(A,
               &global_am,
               &global_an);
    global_mata_m = global_am;
    global_mata_k = global_an;
    if (global_mata_m != global_mata_k) {
        std::cout << "A should be a square matrix." << std::endl;
        std::terminate();
    }

    const PetscInt *row_ownership;
    MatGetOwnershipRanges(A,
                          &row_ownership);

    int band_comm_size, band_comm_rank;
    MPI_Comm_size(device_comm.band_comm,
                  &band_comm_size);
    MPI_Comm_rank(device_comm.band_comm,
                  &band_comm_rank);
#ifndef NDEBUG
    for (int i = 0; i < device_comm.world_size; ++i) {
        MPI_Barrier(MPI_COMM_WORLD);
        if (i == device_comm.world_rank) {
            std::cout << "rank " << i << ": (" << band_comm_size << ", " << band_comm_rank << ")" << std::endl;
        }
    }
#endif
    if (device_comm.owned_device) {
        owned_cpu_start = band_comm_rank * num_band_groups * device_comm.num_tasks_per_device;
        owned_cpu_end = (band_comm_rank + 1) * num_band_groups * device_comm.num_tasks_per_device;
        owned_row_start = row_ownership[owned_cpu_start];
        owned_row_end = row_ownership[owned_cpu_end];
    }

    local_mata_m = owned_row_end - owned_row_start;
    local_mata_k = global_mata_k;

    std::vector<PetscInt> row_idx_vec(local_mata_m,
                                      owned_row_start);
    std::vector<PetscInt> col_idx_vec(local_mata_k - local_mata_m,
                                      0);
    for (int i = 0; i < local_mata_m; ++i) {
        row_idx_vec[i] += i;
    }
    int col_idx = 0;
    for (int i = 0; i < owned_row_start; ++i) {
        col_idx_vec[col_idx] = i;
        col_idx++;
    }
    for (int i = owned_row_end; i < local_mata_k; ++i) {
        col_idx_vec[col_idx] = i;
        col_idx++;
    }
    IS row_idx_petsc_is, col_idx_petsc_is;
    ISCreateGeneral(MPI_COMM_WORLD,
                    local_mata_m,
                    &row_idx_vec[0],
                    PETSC_COPY_VALUES,
                    &row_idx_petsc_is);
    ISCreateGeneral(MPI_COMM_WORLD,
                    local_mata_m,
                    &row_idx_vec[0],
                    PETSC_COPY_VALUES,
                    &wfn_row_is);
    ISCreateGeneral(MPI_COMM_WORLD,
                    local_mata_k - local_mata_m,
                    &col_idx_vec[0],
                    PETSC_COPY_VALUES,
                    &col_idx_petsc_is);
    Mat *local_mat_dense, *local_mat_sparse;
    MatCreateSubMatrices(A,
                         1,
                         &row_idx_petsc_is,
                         &row_idx_petsc_is,
                         MAT_INITIAL_MATRIX,
                         &local_mat_dense);
    MatConvert(local_mat_dense[0],
               MATDENSE,
               MAT_INPLACE_MATRIX,
               &local_mat_dense[0]);
    MatCreateSubMatrices(A,
                         1,
                         &row_idx_petsc_is,
                         &col_idx_petsc_is,
                         MAT_INITIAL_MATRIX,
                         &local_mat_sparse);


    setup_mata_on_device(local_mat_sparse[0],
                         local_mat_dense[0]);

    is_freed = false;

    MatDestroySubMatrices(1,
                          &local_mat_dense);
    MatDestroySubMatrices(1,
                          &local_mat_sparse);
    ISDestroy(&row_idx_petsc_is);
    ISDestroy(&col_idx_petsc_is);
}

void AX::setup_mata_on_device(Mat &A_seq_sparse,
                              Mat &A_seq_dense) {
    MatType seq_type;
    MatGetType(A_seq_sparse,
               &seq_type);
    if (std::string(seq_type) != "seqaij") {
        std::cout << "wrong matrix type: " << seq_type << "passed into AX::setup_mata_on_device, should be MATSEQAIJ."
                  << std::endl;
        std::terminate();
    }

    MatGetType(A_seq_dense,
               &seq_type);
    if (std::string(seq_type) != "seqdense") {
        std::cout << "wrong matrix: " << seq_type << " passed into AX::setup_mata_on_device, should be MATSEQDENSE."
                  << std::endl;
        std::terminate();
    }

    PetscInt sam, sak;
    MatGetSize(A_seq_sparse,
               &sam,
               &sak);

    if (sam != local_mata_m) {
        std::cout << "wrong matrix size m" << std::endl;
        std::terminate();
    } else if (sak != (local_mata_k - local_mata_m)) {
        std::cout << "wrong matrix size n" << std::endl;
        std::terminate();
    }


    MatInfo A_seq_info;
    MatGetInfo(A_seq_sparse,
               MAT_LOCAL,
               &A_seq_info);
    nzs = A_seq_info.nz_used;

    PetscInt parn;
    const PetscInt *ia;
    const PetscInt *ja;
    PetscBool done;
    MatGetRowIJ(A_seq_sparse,
                0,
                PETSC_FALSE,
                PETSC_FALSE,
                &parn,
                &ia,
                &ja,
                &done);
    if (done != PETSC_TRUE) {
        std::cout << "MatGetRowIJ failed." << std::endl;
        std::terminate();
    }
    nrs = parn;
    std::vector<int> ia_h(ia,
                          ia + nrs + 1);
    std::vector<int> ja_h(ja,
                          ja + nzs);
#ifndef NDEBUG
    for (int i = 0; i < nzs; ++i) {
        ja_h[i] += (owned_row_start <= ja_h[i])*gpu_local_dense_m;
    }
#endif
    MatRestoreRowIJ(A_seq_sparse,
                    0,
                    PETSC_FALSE,
                    PETSC_FALSE,
                    &parn,
                    &ia,
                    &ja,
                    &done);
    if (done != PETSC_TRUE) {
        std::cout << "MatRestoreRowIJ failed." << std::endl;
        std::terminate();
    }

    if (device_comm.owned_device) {
        hipError_t cudat_error1, cudat_error2;
        cudat_error1 = hipMalloc(&ia_d,
                                  (nrs + 1) * sizeof(int));
        cudat_error2 = hipMalloc(&ja_d,
                                  nzs * sizeof(int));
        if ((cudat_error1 != hipSuccess) ||
            (cudat_error2 != hipSuccess)) {
            std::cout << "ia_d, ja_d device malloc failed" << std::endl;
            std::terminate();
        }

        cudat_error1 = hipMemcpy(ia_d,
                                  ia_h.data(),
                                  (nrs + 1) * sizeof(int),
                                  hipMemcpyHostToDevice);
        cudat_error2 = hipMemcpy(ja_d,
                                  ja_h.data(),
                                  nzs * sizeof(int),
                                  hipMemcpyHostToDevice);
        if ((cudat_error1 != hipSuccess) ||
            (cudat_error2 != hipSuccess)) {
            std::cout << "ia_d, ja_d device memcpy failed" << std::endl;
            std::terminate();
        }
        adjust_column<<<(nzs + 255) / 256, 256>>>(owned_row_start,
                                                  owned_row_end - owned_row_start,
                                                  nzs,
                                                  ja_d);
    }

    double *a_arr_h;
    MatSeqAIJGetArray(A_seq_sparse,
                      &a_arr_h);

    if (device_comm.owned_device) {
        hipError_t cudat_error1, cudat_error2;
        cudat_error1 = hipMalloc(&a_sparse_arr_d,
                                  nzs * sizeof(double));
        cudat_error2 = hipMemcpy(a_sparse_arr_d,
                                  a_arr_h,
                                  nzs * sizeof(double),
                                  hipMemcpyHostToDevice);
        if ((cudat_error1 != hipSuccess) ||
            (cudat_error2 != hipSuccess)) {
            std::cout << "A_seq_sparse array device malloc/memcpy failed" << std::endl;
            std::terminate();
        }
    }

    MatSeqAIJRestoreArray(A_seq_sparse,
                          &a_arr_h);

    double *a_seq_dense_arr;
    PetscInt dam, dak;
    MatGetSize(A_seq_dense,
               &dam,
               &dak);
    if (dam != dak) {
        std::cout << "non-square size inconsistent for dense mat in AX::setup_mata_on_device." << std::endl;
        std::terminate();
    } else if (dam != local_mata_m) {
        std::cout << "size inconsistent for dense mat m dim in AX::setup_mata_on_device." << std::endl;
        std::terminate();
    }
    MatDenseGetArray(A_seq_dense,
                     &a_seq_dense_arr);

    if (device_comm.owned_device) {
        hipError_t cudat_error1, cudat_error2;
        cudat_error1 = hipMalloc(&a_dense_d,
                                  local_mata_m * local_mata_m * sizeof(double));
        cudat_error2 = hipMemcpy(a_dense_d,
                                  a_seq_dense_arr,
                                  local_mata_m * local_mata_m * sizeof(double),
                                  hipMemcpyHostToDevice);
        if ((cudat_error1 != hipSuccess) ||
            (cudat_error2 != hipSuccess)) {
            std::cout << "A_dense_sparse array device malloc/memcpy failed" << std::endl;
            std::terminate();
        }
    }

    MatDenseRestoreArray(A_seq_dense,
                         &a_seq_dense_arr);
}

void AX::print_matrices() {
    if (is_freed) {
        std::cout << "empty pointer for printing." << std::endl;
        return;
    }

    for (int i = 0; i < device_comm.world_size; ++i) {
        MPI_Barrier(MPI_COMM_WORLD);
        if (i == device_comm.world_rank) {
            if (device_comm.owned_device) {
                std::vector<int> ia(nrs + 1,
                                    0);
                std::vector<int> ja(nzs,
                                    0);
                std::vector<double> arra(nzs,
                                         0);
                std::vector<double> densea(local_mata_m * local_mata_m,
                                           0);
                hipMemcpy(&ia[0],
                           ia_d,
                           (nrs + 1) * sizeof(int),
                           hipMemcpyDeviceToHost);
                hipMemcpy(&ja[0],
                           ja_d,
                           nzs * sizeof(int),
                           hipMemcpyDeviceToHost);
                hipMemcpy(&arra[0],
                           a_sparse_arr_d,
                           nzs * sizeof(double),
                           hipMemcpyDeviceToHost);
                hipMemcpy(&densea[0],
                           a_dense_d,
                           (local_mata_m * local_mata_m) * sizeof(double),
                           hipMemcpyDeviceToHost);
                std::cout << "ia: ";
                for (int i: ia) {
                    std::cout << i << ", ";
                }
                std::cout << std::endl;
                std::cout << "ja: ";
                for (int i: ja) {
                    std::cout << i << ", ";
                }
                std::cout << std::endl;
                std::cout << "arr_a: ";
                for (double i: arra) {
                    std::cout << i << ", ";
                }
                std::cout << std::endl;

                std::cout << "dense: " << std::endl;
                for (int i = 0; i < local_mata_m; ++i) {
                    for (int j = 0; j < local_mata_m; ++j) {
                        std::cout << densea[i + j * local_mata_m] << " ";
                    }
                    std::cout << std::endl;
                }
                std::cout << std::endl;
            }
        }
    }
}

void AX::print_device_double(double *dev,
                             int n,
                             const std::string var_name) const {

    for (int i = 0; i < device_comm.world_size; ++i) {
        MPI_Barrier(MPI_COMM_WORLD);
        if (i == device_comm.world_rank) {
            if (device_comm.owned_device) {
                if (dev == nullptr) {
                    std::cout << "empty pointer for printing." << std::endl;
                    std::terminate();
                }
                std::vector<double> temp(n,
                                         0);
                hipMemcpy(&temp[0],
                           dev,
                           n * sizeof(double),
                           hipMemcpyDeviceToHost);
                std::cout << var_name << ": ";
                for (const auto &i: temp) {
                    std::cout << i << ", ";
                }
                std::cout << std::endl;
            }
        }
    }
}

void AX::print_device_int(int *dev,
                          int n,
                          const std::string var_name) const {
    for (int i = 0; i < device_comm.world_size; ++i) {
        MPI_Barrier(MPI_COMM_WORLD);
        if (i == device_comm.world_rank) {
            if (device_comm.owned_device) {
                if (dev == nullptr) {
                    std::cout << "empty pointer for printing." << std::endl;
                    std::terminate();
                }
                std::vector<int> temp(n,
                                      0);
                hipMemcpy(&temp[0],
                           dev,
                           n * sizeof(int),
                           hipMemcpyDeviceToHost);
                std::cout << var_name << ": ";
                for (const auto &i: temp) {
                    std::cout << i << ", ";
                }
                std::cout << std::endl;
            }
        }
    }
}

void AX::trans_mult(int x_m,
                    int x_n,
                    const double *x,
                    double *ax) const {

    if (x_m != local_mata_m) {
        std::cout << "wrong x dimension for trans_mult" << std::endl;
        std::terminate();
    }

    double *x_d;
    hipError_t cuda_error;
    cuda_error = hipMalloc(&x_d,
                            x_m * x_n * sizeof(double));
    if (cuda_error != hipSuccess) {
        std::cout << "trans_mult malloc x_d failed" << std::endl;
        std::terminate();
    }
    cuda_error = hipMemcpy(x_d,
                            x,
                            x_m * x_n * sizeof(double),
                            hipMemcpyHostToDevice);
    if (cuda_error != hipSuccess) {
        std::cout << "trans_mult memcpy x_d failed" << std::endl;
        std::terminate();
    }

    double *ax_d;
    cuda_error = hipMalloc(&ax_d,
                            local_mata_k * x_n * sizeof(double));
    if (cuda_error != hipSuccess) {
        std::cout << "trans_mult malloc ax_d failed" << std::endl;
        std::terminate();
    }

    hipsparseStatus_t sparse_status;
#ifndef NDEBUG
    //    This part is moved to the contructor to avoid being called multiple times. Kept here for future investigation.
    //    hipsparseHandle_t sparse_handle = 0;
    //    hipsparseMatDescr_t sparse_descr = 0;
    //    hipsparseCreate(&sparse_handle);
    //    sparse_status = hipsparseCreateMatDescr(&sparse_descr);
    //    if (sparse_status != HIPSPARSE_STATUS_SUCCESS) {
    //        std::cout << "Matrix descriptor initialization failed" << std::endl;
    //        std::terminate();
    //    }
    //    hipsparseSetMatType(sparse_descr,
    //                       HIPSPARSE_MATRIX_TYPE_GENERAL);
    //    hipsparseSetMatIndexBase(sparse_descr,
    //                            HIPSPARSE_INDEX_BASE_ZERO);

        print_device_int(ia_d,
                         nrs + 1,
                         "ia_d");
        print_device_int(ja_d,
                         nzs,
                         "ja_d");
        print_device_double(a_sparse_arr_d,
                            nzs,
                            "a_sparse_arr_d");
        print_device_double(x_d,
                            x_m * x_n,
                            "x_d");
        std::cout << local_mata_m << ", " << x_n << ", " << local_mata_k << std::endl;
#endif
    sparse_status = hipsparseDcsrmm(sparse_handle,
                                   HIPSPARSE_OPERATION_TRANSPOSE,
                                   local_mata_m,
                                   x_n,
                                   local_mata_k,
                                   nzs,
                                   &double_one,
                                   sparse_descr,
                                   a_sparse_arr_d,
                                   ia_d,
                                   ja_d,
                                   x_d,
                                   local_mata_m,
                                   &double_zero,
                                   ax_d,
                                   local_mata_k);
    if (sparse_status != HIPSPARSE_STATUS_SUCCESS) {
        std::cout << "CUSPARSE matrix-matrix multiplication failed with status: " << sparse_status << std::endl;
        std::terminate();
    }

#ifndef NDEBUG
    print_device_double(ax_d,
                        local_mata_k * x_n,
                        "ax_d");


//    This part is moved to the contructor to avoid being called multiple times. Kept here for future investigation.
//    sparse_status = hipsparseDestroyMatDescr(sparse_descr);
//    sparse_descr = 0;
//    if (sparse_status != HIPSPARSE_STATUS_SUCCESS) {
//        std::cout << "Matrix descriptor destruction failed" << std::endl;
//        std::terminate();
//    }
//
//    sparse_status = hipsparseDestroy(sparse_handle);
//    sparse_handle = 0;
//    if (sparse_status != HIPSPARSE_STATUS_SUCCESS) {
//        std::cout << "CUSPARSE Library release of resources failed" << std::endl;
//        std::terminate();
//    }

//    hipblasStatus_t blas_stat;
//    hipblasHandle_t blas_handle = 0;
//
//    blas_stat = hipblasCreate(&blas_handle);
//    if (blas_stat != HIPBLAS_STATUS_SUCCESS) {
//        std::cout << "CUBLAS Library create handle failed" << std::endl;
//        std::terminate();
//    }

    print_device_double(a_dense_d,
                        local_mata_m * local_mata_m,
                        "a_dense_d");
#endif

    hipblasStatus_t blas_stat;
    blas_stat = hipblasDgemm(blas_handle,
                            HIPBLAS_OP_T,
                            HIPBLAS_OP_N,
                            local_mata_m,
                            x_n,
                            local_mata_m,
                            &double_one,
                            a_dense_d,
                            local_mata_m,
                            x_d,
                            local_mata_m,
                            &double_one,
                            &ax_d[owned_row_start],
                            local_mata_k);
    if (blas_stat != HIPBLAS_STATUS_SUCCESS) {
        std::cout << "CUBLAS matrix-matrix multiplication failed" << std::endl;
        std::terminate();
    }

// code to transpose result, save for future use.
//    double *axt_d;
//    cuda_error = hipMalloc(&axt_d,
//                            local_mata_k * x_n * sizeof(double));
//    if (cuda_error != hipSuccess) {
//        std::cout << "trans_mult malloc axt_d failed" << std::endl;
//        std::terminate();
//    }
//
//    blas_stat = hipblasDgeam(blas_handle,
//                            HIPBLAS_OP_T,
//                            HIPBLAS_OP_N,
//                            x_n,
//                            local_mata_k,
//                            &double_one,
//                            ax_d,
//                            local_mata_k,
//                            &double_zero,
//                            axt_d,
//                            x_n,
//                            axt_d,
//                            x_n);
//    if (blas_stat != HIPBLAS_STATUS_SUCCESS) {
//        std::cout << "CUBLAS matrix dgeam failed" << std::endl;
//        std::terminate();
//    }

    cuda_error = hipMemcpy(ax,
                            ax_d,
                            local_mata_k * x_n * sizeof(double),
                            hipMemcpyDeviceToHost);
    if (cuda_error != hipSuccess) {
        std::cout << "trans_mult memcpy axt_d to host failed" << std::endl;
        std::terminate();
    }

    hipFree(x_d);
//    hipFree(axt_d);
    hipFree(ax_d);
}

void AX::free_mata_on_device() {
    hipFree(ia_d);
    hipFree(ja_d);
    hipFree(a_sparse_arr_d);
    hipFree(a_dense_d);
    is_freed = true;
}

AX::~AX() {

    if (is_freed != true) {
        free_mata_on_device();
    }
    if (device_comm.owned_device) {
        hipsparseStatus_t sparse_status;
        sparse_status = hipsparseDestroyMatDescr(sparse_descr);
        sparse_descr = 0;
        if (sparse_status != HIPSPARSE_STATUS_SUCCESS) {
            std::cout << "Matrix descriptor destruction failed" << std::endl;
            std::terminate();
        }

        sparse_status = hipsparseDestroy(sparse_handle);
        sparse_handle = 0;
        if (sparse_status != HIPSPARSE_STATUS_SUCCESS) {
            std::cout << "CUSPARSE Library release of resources failed" << std::endl;
            std::terminate();
        }

        hipblasStatus_t blas_stat;
        blas_stat = hipblasDestroy(blas_handle);
        blas_handle = 0;
        if (blas_stat != HIPBLAS_STATUS_SUCCESS) {
            std::cout << "CUBLAS Library release of resources failed" << std::endl;
            std::terminate();
        }
    }
    int mpi_finalized;
    MPI_Finalized(&mpi_finalized);
    if (!mpi_finalized) {
        ISDestroy(&wfn_row_is);
        ISDestroy(&wfn_col_is);
    }
    is_freed = true;
}

void AX::perform_ax(Mat &X,
                    Mat &AX) {

    Mat *sub_X;
    MatCreateSubMatrices(X,
                         1,
                         &wfn_row_is,
                         &wfn_col_is,
                         MAT_INITIAL_MATRIX,
                         &sub_X);
    int x_m = local_mata_m;
    int x_n = owned_wfn_end - owned_wfn_start;
    double *sub_X_ptr;
    MatDenseGetArray(sub_X[0],
                     &sub_X_ptr);

#ifndef NDEBUG
    for (int i = 0; i < device_comm.world_size; ++i) {
        MPI_Barrier(MPI_COMM_WORLD);
        if (i == device_comm.world_rank) {
            std::cout << "rank " << i << ": " << std::endl;
            MatView(sub_X[0],
                    PETSC_VIEWER_STDOUT_SELF);
        }
    }

    Mat AsubX;
    MatCreateSeqDense(MPI_COMM_SELF,
                      local_mata_k,
                      x_n,
                      PETSC_NULL,
                      &AsubX);
    double *AsubX_ptr;
    MatDenseGetArray(AsubX,
                     &AsubX_ptr);
    if (device_comm.owned_device) {
        trans_mult(x_m,
                   x_n,
                   sub_X_ptr,
                   AsubX_ptr);
    }
    MatDenseRestoreArray(sub_X[0],
                         &sub_X_ptr);
    MPI_Allreduce(MPI_IN_PLACE,
                  AsubX_ptr,
                  local_mata_k * x_n,
                  MPI_DOUBLE,
                  MPI_SUM,
                  device_comm.band_comm);

    for (int i = 0; i < device_comm.world_size; ++i) {
        MPI_Barrier(MPI_COMM_WORLD);
        if (i == device_comm.world_rank) {
            std::cout << "rank " << i << ": " << std::endl;
            MatView(AsubX,
                    PETSC_VIEWER_STDOUT_SELF);
        }
    }
    MatDenseRestoreArray(AsubX,
                         &AsubX_ptr);
#endif

    double *A_subX = (double *) malloc(local_mata_k * x_n * sizeof(double));

    if (device_comm.owned_device) {
        trans_mult(x_m,
                   x_n,
                   sub_X_ptr,
                   A_subX);
    }

    MatDenseRestoreArray(sub_X[0],
                         &sub_X_ptr);
    MatDestroySubMatrices(1,
                          &sub_X);
    MPI_Allreduce(MPI_IN_PLACE,
                  A_subX,
                  local_mata_k * x_n,
                  MPI_DOUBLE,
                  MPI_SUM,
                  device_comm.band_comm);

#ifndef NDEBUG
    for (int irank = 0; irank < device_comm.world_size; ++irank) {
        MPI_Barrier(MPI_COMM_WORLD);
        if (irank == device_comm.world_rank && device_comm.owned_device == true) {
            std::cout << "rank: " << irank << std::endl;
            for (int i = 0; i < local_mata_k; ++i) {
                for (int j = 0; j < x_n; ++j) {
                    printf("%.6e ", A_subX[i + j*local_mata_k]);
                }
                std::cout << std::endl;
            }
        }
    }
#endif

//    double *owned_Asub_X = (double *) malloc(x_m*x_n*sizeof(double));



}

namespace {
    __global__ void adjust_column(int owned_row_start,
                                  int owned_row_size,
                                  int n,
                                  int *col_idx) {

        int index = blockIdx.x * blockDim.x + threadIdx.x;
        int stride = blockDim.x * gridDim.x;
        for (int i = index; i < n; i += stride) {
            col_idx[i] += (owned_row_start <= col_idx[i]) * owned_row_size;
        }
    }
}