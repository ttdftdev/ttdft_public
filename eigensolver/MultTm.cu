/******************************************************************************
 * Copyright (c) 2020-2021.                                                   *
 * The Regents of the University of Michigan and TTDFT authors.               *
 *                                                                            *
 * This file is part of the TTDFT code.                                       *
 *                                                                            *
 * TTDFT is free software: you can redistribute it and/or modify              *
 *  it under the terms of the Lesser GNU General Public License as            *
 *  published by the Free Software Foundation, either version 3 of            *
 *  the License, or (at your option) any later version.                       *
 *                                                                            *
 *  TTDFT is distributed in the hope that it will be useful, but              *
 *  WITHOUT ANY WARRANTY; without even the implied warranty                   *
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.                      *
 *  See the Lesser GNU General Public License for more details.               *
 *                                                                            *
 * You should have received a copy of the GNU Lesser General Public           *
 * License at the top level of TTDFT distribution.  If not, see               *
 * <https://www.gnu.org/licenses/>.                                           *
 ******************************************************************************/

//
// Created by iancclin
//

#include "MultTm.h"
#include "DeviceUtils.cuh"
#include <hip/hip_runtime.h>
#include <iostream>

MultTM::MultTM() {

  MPI_Comm nodal_comm;
  int nodal_size, nodal_rank;
  MPI_Comm_split_type(MPI_COMM_WORLD, MPI_COMM_TYPE_SHARED, 0, MPI_INFO_NULL,
                      &nodal_comm);

  MPI_Comm_size(nodal_comm, &nodal_size);
  MPI_Comm_rank(nodal_comm, &nodal_rank);
  int max_nodal_size, min_nodal_size;
  MPI_Allreduce(&nodal_size,
                &max_nodal_size,
                1,
                MPI_INT,
                MPI_MAX,
                MPI_COMM_WORLD);
  MPI_Allreduce(&nodal_size,
                &min_nodal_size,
                1,
                MPI_INT,
                MPI_MIN,
                MPI_COMM_WORLD);

  if (max_nodal_size!=min_nodal_size) {
    std::cout << "ERROR: each node should have the same number of cpus."
              << std::endl;
    std::terminate();
  }

  device_utils::device_get_device_count(local_num_devices);
  if (local_num_devices==0) {
    std::cout << "ERROR: no gpu or gpu is more than cpus on the node "
                 "(currently not supported.)"
              << std::endl;
    std::terminate();
  }

  MPI_Comm_size(MPI_COMM_WORLD, &world_size);
  MPI_Comm_rank(MPI_COMM_WORLD, &world_rank);
  int num_tasks_per_device = nodal_size/local_num_devices;
  int local_device_id = nodal_rank/num_tasks_per_device;
  device_utils::device_set_device(local_device_id);
  global_num_devices = local_num_devices*(world_size/nodal_size);
  int global_device_id = world_rank/num_tasks_per_device;
  MPI_Comm_split(MPI_COMM_WORLD, global_device_id, world_rank, &comm_tm);

  MPI_Comm_size(comm_tm, &comm_tm_size);
  MPI_Comm_rank(comm_tm, &comm_tm_rank);

  hipblasCreate(&cublas_handle);
  hipsolverDnCreate(&cusolverdn_handle);
  hipStreamCreateWithFlags(&cusolver_stream, hipStreamNonBlocking);
  hipsolverSetStream(cusolverdn_handle, cusolver_stream);
}

void MultTM::mult(Mat &X, Mat &Y, std::vector<double> &result) {

  const PetscInt *x_range, *y_range;
  MatGetOwnershipRanges(X, &x_range);
  MatGetOwnershipRanges(Y, &y_range);
  for (int i_rank = 0; i_rank < world_size; ++i_rank) {
    if (x_range[i_rank]!=y_range[i_rank]) {
      std::cout << "X and Y matrix should have same row ownerships."
                << std::endl;
      std::terminate();
    }
  }

  PetscInt global_m, global_n;
  MatGetSize(X, &global_m, &global_n);
  std::vector<PetscInt> row_t, col_t(global_n, 0);
  for (int i = 0; i < global_n; ++i)
    col_t[i] = i;
  if (comm_tm_rank==0) {
    int start = x_range[world_rank];
    int end = x_range[world_rank + comm_tm_size];
    row_t = std::vector<PetscInt>(end - start, start);
    for (int i = 0; i < row_t.size(); ++i)
      row_t[i] += i;
  }

  IS is_row, is_col;
  ISCreateGeneral(MPI_COMM_WORLD,
                  row_t.size(),
                  row_t.data(),
                  PETSC_COPY_VALUES,
                  &is_row);
  ISCreateGeneral(MPI_COMM_WORLD,
                  col_t.size(),
                  col_t.data(),
                  PETSC_COPY_VALUES,
                  &is_col);

  Mat *subX, *subY;
  MatCreateSubMatrices(X,
                       1,
                       &is_row,
                       &is_col,
                       MAT_INITIAL_MATRIX,
                       &subX);
  MatCreateSubMatrices(Y,
                       1,
                       &is_row,
                       &is_col,
                       MAT_INITIAL_MATRIX,
                       &subY);

  result = std::vector<double>(col_t.size()*col_t.size(), 0.0);

  if (comm_tm_rank==0) {
    double *x_lht, *y_lht;
    MatDenseGetArray(subX[0], &x_lht);
    MatDenseGetArray(subY[0], &y_lht);

    double *x_ld, *y_ld, *xty_ld;
    hipError_t cudat_error1, cudat_error2, cudat_error3;
    cudat_error1 =
        hipMalloc(&x_ld,
                   row_t.size()*col_t.size()*sizeof(double));
    cudat_error2 =
        hipMalloc(&y_ld,
                   row_t.size()*col_t.size()*sizeof(double));
    cudat_error3 =
        hipMalloc(&xty_ld,
                   col_t.size()*col_t.size()*sizeof(double));

    if ((cudat_error1!=hipSuccess) || (cudat_error2!=hipSuccess) ||
        (cudat_error3!=hipSuccess)) {
      std::cout << "device malloc failed." << std::endl;
      std::terminate();
    }

    cudat_error1 =
        hipMemcpy(&x_ld, &x_lht, row_t.size()*col_t.size()*sizeof(double),
                   hipMemcpyHostToDevice);
    cudat_error2 =
        hipMemcpy(&y_ld, &y_lht, row_t.size()*col_t.size()*sizeof(double),
                   hipMemcpyHostToDevice);
    if ((cudat_error1!=hipSuccess) || (cudat_error2!=hipSuccess)) {
      std::cout << "host to device memory copy failed." << std::endl;
      std::terminate();
    }

    double double_one = 1.0;
    hipblasStatus_t blas_stat;
    blas_stat =
        hipblasDgemm(cublas_handle,
                    HIPBLAS_OP_T,
                    HIPBLAS_OP_N,
                    col_t.size(),
                    col_t.size(),
                    row_t.size(),
                    &double_one,
                    x_ld,
                    row_t.size(),
                    y_ld,
                    row_t.size(),
                    &double_one,
                    xty_ld,
                    col_t.size());
    if (blas_stat!=HIPBLAS_STATUS_SUCCESS) {
      std::cout << "CUBLAS matrix-matrix multiplication failed" << std::endl;
      std::terminate();
    }

    cudat_error1 = hipFree(&x_ld);
    cudat_error2 = hipFree(&y_ld);

    hipMemcpy(result.data(), xty_ld,
               col_t.size()*col_t.size()*sizeof(double),
               hipMemcpyDeviceToHost);

    cudat_error3 = hipFree(&xty_ld);

    if ((cudat_error1!=hipSuccess) || (cudat_error2!=hipSuccess) ||
        (cudat_error3!=hipSuccess)) {
      std::cout << "device memory free failed." << std::endl;
      std::terminate();
    }

    MatDenseRestoreArray(subX[0], &x_lht);
    MatDenseRestoreArray(subY[0], &y_lht);
  }

  MatDestroySubMatrices(1, &subX);
  MatDestroySubMatrices(1, &subY);

  MPI_Allreduce(MPI_IN_PLACE,
                result.data(),
                result.size(),
                MPI_DOUBLE,
                MPI_SUM,
                MPI_COMM_WORLD);
}

void MultTM::orth(Mat &X) {
  std::vector<double> S;
  mult(X, X, S);
  PetscInt global_m, global_n;
  MatGetSize(X, &global_m, &global_n);
  int mat_m = global_m, mat_n = global_n;

  hipError_t cudat_error1, cudat_error2;
  int *info_d = nullptr;
  int worksize_d = 0;
  double *workspace_d = nullptr;

  double *S_d = nullptr;
  cudat_error1 = hipMalloc(&S_d, sizeof(double)*S.size());
  if ((cudat_error1!=hipSuccess)) {
    std::cout << "device memory allocation failed." << std::endl;
    std::terminate();
  }

  cudat_error1 = hipMemcpy(S_d,
                            S.data(),
                            sizeof(double)*S.size(),
                            hipMemcpyHostToDevice);
  if ((cudat_error1!=hipSuccess)) {
    std::cout << "device memory copy failed." << std::endl;
    std::terminate();
  }

  hipsolverStatus_t solver_stat;
  hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;
  solver_stat = hipsolverDnDpotrf_bufferSize(cusolverdn_handle,
                                            uplo,
                                            mat_m,
                                            S_d,
                                            mat_m,
                                            &worksize_d);
  if (solver_stat!=HIPSOLVER_STATUS_SUCCESS) {
    std::cout << "CUSOLVER buffer preparation failed" << std::endl;
    std::terminate();
  }

  cudat_error1 = hipMalloc(&workspace_d,
                            sizeof(double)*worksize_d);
  cudat_error2 = hipMalloc(&info_d,
                            sizeof(int));
  if ((cudat_error1!=hipSuccess) || (cudat_error2!=hipSuccess)) {
    std::cout << "device memory allocation failed." << std::endl;
    std::terminate();
  }
  cudat_error1 = hipMemset(&info_d,
                            0,
                            sizeof(int));
  if ((cudat_error1!=hipSuccess)) {
    std::cout << "device memory set failed." << std::endl;
    std::terminate();
  }

  solver_stat = hipsolverDnDpotrf(cusolverdn_handle,
                                 uplo,
                                 mat_m,
                                 S_d,
                                 mat_m,
                                 workspace_d,
                                 worksize_d,
                                 info_d);
  if (solver_stat!=HIPSOLVER_STATUS_SUCCESS) {
    std::cout << "CUSOLVER potrf failed" << std::endl;
    std::terminate();
  }
  cudat_error1 = hipFree(workspace_d);
  if ((cudat_error1!=hipSuccess)) {
    std::cout << "device memory free failed." << std::endl;
    std::terminate();
  }


  PetscInt local_m, local_n;
  MatGetSize(X, &local_m, &local_n);
  int local_mat_m = local_m, local_mat_n = local_n;

  double *X_lh, *X_ld;
  MatDenseGetArray(X, &X_lh);

#ifdef SOLVE_WITH_INVERSE
  solver_stat = cusolverDnDtrtri_bufferSize(cusolverdn_handle,
                                            uplo,
                                            HIPBLAS_DIAG_NON_UNIT,
                                            mat_m,
                                            S_d,
                                            mat_m,
                                            &worksize_d);
  if (solver_stat!=HIPSOLVER_STATUS_SUCCESS) {
    std::cout << "CUSOLVER trtri buffer preparation failed" << std::endl;
    std::terminate();
  }

  cudat_error1 = hipMalloc(&workspace_d,
                            sizeof(double)*worksize_d);
  if ((cudat_error1!=hipSuccess)) {
    std::cout << "device memory allocation failed." << std::endl;
    std::terminate();
  }

  cudat_error1 = hipMemset(&info_d,
                            0,
                            sizeof(int));
  if ((cudat_error1!=hipSuccess)) {
    std::cout << "device memory set failed." << std::endl;
    std::terminate();
  }

  solver_stat = cusolverDnDtrtri(cusolverdn_handle,
                                 uplo,
                                 HIPBLAS_DIAG_NON_UNIT,
                                 mat_m,
                                 S_d,
                                 mat_m,
                                 workspace_d,
                                 worksize_d,
                                 info_d);
  if (solver_stat!=HIPSOLVER_STATUS_SUCCESS) {
    std::cout << "CUSOLVER trtri execution failed" << std::endl;
    std::terminate();
  }
  cudat_error1 = hipFree(workspace_d);
  if ((cudat_error1!=hipSuccess)) {
    std::cout << "device memory free failed." << std::endl;
    std::terminate();
  }
#endif

  cudat_error1 = hipMalloc(&X_ld,
                            local_mat_m*local_mat_n*sizeof(double));
  if ((cudat_error1!=hipSuccess)) {
    std::cout << "device memory allocation failed." << std::endl;
    std::terminate();
  }

  cudat_error1 =
      hipMemcpy(&X_ld, &X_lh,
                 local_mat_m*local_mat_n*sizeof(double),
                 hipMemcpyHostToDevice);
  if ((cudat_error1!=hipSuccess)) {
    std::cout << "device memory copy failed." << std::endl;
    std::terminate();
  }

  double double_one = 1.0;
  hipblasStatus_t blas_stat;

#ifdef SOLVE_WITH_INVERSE
  double *XLt_d;
  cudat_error1 = hipMalloc(&XLt_d,
                            local_mat_m*local_mat_n*sizeof(double));
  if ((cudat_error1!=hipSuccess)) {
    std::cout << "device memory allocation failed." << std::endl;
    std::terminate();
  }

  blas_stat =
      hipblasDtrmm(cublas_handle,
                  HIPBLAS_SIDE_RIGHT,
                  uplo,
                  HIPBLAS_OP_T,
                  HIPBLAS_DIAG_NON_UNIT,
                  local_mat_m,
                  local_mat_n,
                  &double_one,
                  S_d,
                  local_mat_n,
                  X_ld,
                  local_mat_m,
                  XLt_d,
                  local_mat_m);
  if (blas_stat!=HIPBLAS_STATUS_SUCCESS) {
    std::cout << "CUBLAS triangular matrix-matrix multiplication failed" << std::endl;
    std::terminate();
  }

  cudat_error1 =
      hipMemcpy(&X_lh, &XLt_d,
                 local_mat_m*local_mat_n*sizeof(double),
                 hipMemcpyDeviceToHost);
  if ((cudat_error1!=hipSuccess)) {
    std::cout << "host memory copy failed." << std::endl;
    std::terminate();
  }


  cudat_error1 = hipFree(&XLt_d);
  if ((cudat_error1!=hipSuccess)) {
  std::cout << "device memory free failed." << std::endl;
  std::terminate();
  }
#else

  blas_stat =
      hipblasDtrsm(cublas_handle,
                  HIPBLAS_SIDE_RIGHT,
                  uplo,
                  HIPBLAS_OP_T,
                  HIPBLAS_DIAG_NON_UNIT,
                  local_mat_m,
                  local_mat_n,
                  &double_one,
                  S_d,
                  local_mat_n,
                  X_ld,
                  local_mat_m);
  if (blas_stat!=HIPBLAS_STATUS_SUCCESS) {
    std::cout << "CUBLAS triangular matrix-matrix solve failed" << std::endl;
    std::terminate();
  }

  cudat_error1 =
      hipMemcpy(&X_lh, &X_ld,
                 local_mat_m*local_mat_n*sizeof(double),
                 hipMemcpyDeviceToHost);
  if ((cudat_error1!=hipSuccess)) {
    std::cout << "host memory copy failed." << std::endl;
    std::terminate();
  }

#endif

  cudat_error1 = hipFree(info_d);
  if ((cudat_error1!=hipSuccess)) {
    std::cout << "device memory free failed." << std::endl;
    std::terminate();
  }

  cudat_error1 = hipFree(&S_d);
  cudat_error2 = hipFree(&X_ld);


  if ((cudat_error1!=hipSuccess) || (cudat_error2!=hipSuccess)) {
    std::cout << "device memory free failed." << std::endl;
    std::terminate();
  }

  MatDenseRestoreArray(X, &X_lh);
}

MultTM::~MultTM() {
  hipsolverDnDestroy(cusolverdn_handle);
  hipStreamDestroy(cusolver_stream);
  hipblasDestroy(cublas_handle);
}
